#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <omp.h>
#include <string.h>
#include <hip/hip_runtime.h>

#include "../include/utils.h"

#define NUM_BLOCKS 128
#define NUM_THREADS_PER_BLOCK 256
#define SIZE NUM_BLOCKS*NUM_THREADS_PER_BLOCK


//return is number of iterarions
__global__ void k_means(float *cluster_x,float *cluster_y,const float *arr_x,const float *arr_y,int *points,const int N,const int K){
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if(id < N){
		int lid = threadIdx.x; 

		//__shared__ int points[NUM_THREADS_PER_BLOCK]; 

		__shared__ float mean_x[10],mean_y[10]; //keep values to calculate new centroid

		//__shared__ int local_elem_cluster[K];

		__shared__ float local_arr_x[NUM_THREADS_PER_BLOCK],local_arr_y[NUM_THREADS_PER_BLOCK];

		local_arr_x[lid] = arr_x[id];
		local_arr_y[lid] = arr_y[id];

		//reset arrays
		for(int i = 0; i < K;i++){
				mean_x[i] = 0;	
				mean_y[i] = 0;
				//local_elem_cluster[i] = 0;
		}



		
		float dist[10]; 
		int ind = 0; //lower distance index
		for(int i = 0; i < K;i++){//calculate distance between points and centroids 
			float x = (cluster_x[i] - local_arr_x[lid]);
			float y = (cluster_y[i] - local_arr_y[lid]);
			dist[i]  = x * x;  
			dist[i] += y * y;
			DEBUG("cluster_x[%d]=%f | cluster_y[%d]=%f | local_arr_x[%d]=%f | local_arr_x[%d]=%f | dist[%d]=%f\n",i,cluster_x[i],i,cluster_y[i],i,local_arr_x[i],i,local_arr_y[i],i,dist[i]);
		}

		//find minimum value for the distance between centroid and point
		for(int i = 1; i < K;i++){
			ind = (dist[i] < dist[ind])?i:ind;
		}
		
		DEBUG("ind:%d\n",ind);

		points[id] = ind; // assigns the new lowest distance centroid to the point 








		/*
		if(threadIdx.x == 0){
			for(int i = 0; i < NUM_THREADS_PER_BLOCK; i++){
				int ind = points[i];

				mean_x[ind] += local_arr_x[i]; // add this point to the sum of points belonging to cluster
				mean_y[ind] += local_arr_y[i];
				local_elem_cluster[ind]++; //update number of elements in cluster
			}

			for(int i = 0;i < K;i++){

			}
		}*/
	}
}

int launch_kernel(float *cluster_x,float *cluster_y,const float *arr_x,const float *arr_y,int *n_elem_cluster,const int N,const int K,const int T){

	int ret = 0;

	const int size_clusters = K * sizeof(float);
	const int size_points = N * sizeof(float);
	int *points = (int*)malloc(N * sizeof(int));
	float mean_x[K],mean_y[K];

	float *d_cluster_x;
	float *d_cluster_y;
	float *d_arr_x;
	float *d_arr_y;
	int *d_points;
	int *d_n_elem_cluster;

	//allocate space in the device
	hipMalloc((void**) &d_cluster_x, size_clusters);
	hipMalloc((void**) &d_cluster_y, size_clusters);
	hipMalloc((void**) &d_arr_x, size_points);
	hipMalloc((void**) &d_arr_y, size_points);
	//hipMalloc((void**) &d_n_elem_cluster, K * sizeof(int));
	hipMalloc((void**) &d_points, N * sizeof(int));

	//copy data from host to device
	hipMemcpy (d_cluster_x,cluster_x,size_clusters,hipMemcpyHostToDevice);
	hipMemcpy (d_cluster_y,cluster_y,size_clusters,hipMemcpyHostToDevice);
	hipMemcpy (d_arr_x,arr_x,size_points,hipMemcpyHostToDevice);
	hipMemcpy (d_arr_y,arr_y,size_points,hipMemcpyHostToDevice);

	//call Kernel
	for(int ite = 0; ite < 20;ite++){
		startKernelTime ();
		k_means <<< N/NUM_THREADS_PER_BLOCK + 1, NUM_THREADS_PER_BLOCK >>> (d_cluster_x,d_cluster_y,d_arr_x,d_arr_y,d_points,N,K);
		stopKernelTime ();
		hipMemcpy (points,d_points, N * sizeof(int),hipMemcpyDeviceToHost);

		for(int i = 0;i < K;i++){
			mean_x[i] = 0; // add this point to the sum of points belonging to cluster
			mean_y[i] = 0;
			n_elem_cluster[i] = 0;
		}

		for(int i = 0; i < N; i++){
			int ind = points[i];
			DEBUG("point[%d]=%d\n",i,ind);
			mean_x[ind] += arr_x[i]; // add this point to the sum of points belonging to cluster
			mean_y[ind] += arr_y[i];
			n_elem_cluster[ind]++; //update number of elements in cluster

		}

		for(int i = 0; i < K;i++){
			cluster_x[i] = mean_x[i] / (n_elem_cluster[i]);
			cluster_y[i] = mean_y[i] / (n_elem_cluster[i]);
			DEBUG("new cluster_x[%d]=%f | cluster_y[%d]=%f | mean_x[%d]=%f |  mean_y[%d]=%f | elem[%d]=%d\n",i,cluster_x[i],i,cluster_y[i],i,mean_x[i],i,mean_y[i],i,n_elem_cluster[i]);
		}

		hipMemcpy (d_cluster_x,cluster_x,size_clusters,hipMemcpyHostToDevice);
		hipMemcpy (d_cluster_y,cluster_y,size_clusters,hipMemcpyHostToDevice);

		ret++;
	}

	//Retrive information from device
	//hipMemcpy (cluster_x,d_cluster_x,size_clusters,hipMemcpyDeviceToHost);
	//hipMemcpy (cluster_y,d_cluster_y,size_clusters,hipMemcpyDeviceToHost);
	//hipMemcpy (n_elem_cluster,d_n_elem_cluster, K * sizeof(int),hipMemcpyDeviceToHost);

	return ret;
}

int main(int argc, char const *argv[]){
	int N = atoi(argv[1]);
	int K = atoi(argv[2]);
	int T = atoi(argv[3]);

	float *arr_x,*arr_y;
	float cluster_x[K],cluster_y[K];
	int n_elem_cluster[K];

	arr_x = (float*) malloc(sizeof(float) * N);
	arr_y = (float*) malloc(sizeof(float) * N);

	init(N,K,arr_x,arr_y,cluster_x,cluster_y);

	int iterarion = launch_kernel(cluster_x,cluster_y,arr_x,arr_y,n_elem_cluster,N,K,T);

	free(arr_x);
	free(arr_y);

	print_ret(cluster_x,cluster_y,n_elem_cluster,N,K,iterarion);
	return 0;
}